#include "hip/hip_runtime.h"
#include <iostream>
#include <limits>
#include <cstdlib>
#include <iostream>


#include "gpu.hpp"
#include "cutil_inline.h"
#include "hiprand.h"
#include "const.h"

const unsigned int ANT_COUNT = 512;
const unsigned int MAX_STEPS = 100;

const float IMPORTANCE_OF_PHEROMONE = 5;
const float PHEROMONE_EVAPORATION = 0.2;
const float TOTAL_PHEROMONE_FOR_TRAIL = 100;

size_t paths_gpu[ANT_COUNT][MAX_STEPS];

void generateRandomNumbers(float *a, size_t amount){
    for(int i = 0; i < amount; i++){
        a[i] = (float)rand()/(float)(RAND_MAX);
    }
}


__device__
int selectFromDistribution(float *probabilityDistribution, float r){
    float sum = 0;
    for(int i = 0; i < NODE_COUNT; i++){
        sum += probabilityDistribution[i];
        if(r < sum){
            return i;
        }
    }

    return 0;
}

//randomNumbers should be an array of size MAX_STEPS * ANT_COUNT populated using generateRandomNumbers
__global__
void update_ant_positions(size_t goal, size_t *paths, float *weights, float *graph, float *randomNumbers, size_t start)
{
    const size_t antIndex = blockIdx.x * blockDim.x + threadIdx.x;
    paths[antIndex * MAX_STEPS] = start;
    for(int step = 0; step < MAX_STEPS - 1; step++){
        int currentPathIndex = (antIndex * MAX_STEPS) + step;
        int movementLocation = paths[currentPathIndex];
        if(paths[currentPathIndex] != goal){
            float probabilityOfAntMovingToNode[NODE_COUNT];
            for(int nodeIndex = 0; nodeIndex < NODE_COUNT; nodeIndex++){
                probabilityOfAntMovingToNode[nodeIndex] = weights[paths[currentPathIndex] * NODE_COUNT + nodeIndex] * IMPORTANCE_OF_PHEROMONE;
                //Previously visited causes divergence which causes timeout due to poor performance
            }

            float sumOfWeights = 0;
            for(int i = 0; i < NODE_COUNT; i++){
                sumOfWeights += probabilityOfAntMovingToNode[i];
            }

            for(int i = 0; i < NODE_COUNT; i++){
                probabilityOfAntMovingToNode[i] /= sumOfWeights;
            }

            movementLocation = selectFromDistribution(probabilityOfAntMovingToNode, randomNumbers[(step * ANT_COUNT) + antIndex]);

        }
        paths[currentPathIndex + 1] = movementLocation;
    }

}

//BLOCK_SIZE * GRID_SIZE should equal nodeCount squared (Size of weight array)
__global__
void pheromone_evaporation(float *weights)
{
    const size_t offset = blockIdx.x * blockDim.x + threadIdx.x;
    weights[offset] *= PHEROMONE_EVAPORATION;
}

//BLOCK_SIZE * GRID_SIZE should equal ANT_COUNT
__global__
void path_lengths(size_t goal, size_t *paths, size_t *path_lengths)
{
    const size_t antIndex = blockIdx.x * blockDim.x + threadIdx.x;
    path_lengths[antIndex] = 0;
    for (unsigned int j = 0; j < MAX_STEPS; j++) {
        if (paths[antIndex * MAX_STEPS + j] != goal) {
            path_lengths[antIndex]++;
        }
    }
}


void update_gpu(float (*weights)[NODE_COUNT][NODE_COUNT], float (*graph)[NODE_COUNT][NODE_COUNT], size_t goal, size_t nc, size_t start){
    int devID;
    hipDeviceProp_t props;

    cutilSafeCall(hipGetDevice(&devID));
    cutilSafeCall(hipGetDeviceProperties(&props, devID));

    // calculate the storage requirements of each array
    const size_t paths_length = ANT_COUNT * MAX_STEPS * sizeof(size_t),
    weights_length = nc * nc * sizeof(float),
    graph_length = nc * nc * sizeof(float),
    path_lengths_length = ANT_COUNT * sizeof(size_t),
    random_numbers_length = ANT_COUNT * MAX_STEPS * sizeof(float);

    std::cout << "Path length: " << paths_length << " bytes" << std::endl
    << "Weights length: " << weights_length << " bytes" << std::endl
    << "Graph length: " << graph_length << " bytes" << std::endl
    << "Path lengths length: " << path_lengths_length << " bytes" << std::endl;

    // allocate memory on device
    float *d_weights, *d_graph, *d_random_numbers;
    size_t *d_paths, *d_path_lengths;
    cutilSafeCall(hipMalloc(&d_paths, paths_length));
    cutilSafeCall(hipMalloc(&d_weights, weights_length));
    cutilSafeCall(hipMalloc(&d_graph, graph_length));
    cutilSafeCall(hipMalloc(&d_path_lengths, path_lengths_length));
    cutilSafeCall(hipMalloc(&d_random_numbers, random_numbers_length));


    // copy data to device
    cutilSafeCall(hipMemcpy(d_paths, &paths_gpu, paths_length, hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy(d_weights, &(*weights), weights_length, hipMemcpyHostToDevice));
    cutilSafeCall(hipMemcpy(d_graph, &(*graph), graph_length, hipMemcpyHostToDevice));


    float h_random_numbers[random_numbers_length];
    generateRandomNumbers(h_random_numbers, random_numbers_length);
    cutilSafeCall(hipMemcpy(d_random_numbers, &h_random_numbers, random_numbers_length, hipMemcpyHostToDevice));

    // 0-initialise the path lengths array on the device
    cutilSafeCall(hipMemset(d_path_lengths, 0, path_lengths_length));

    // calculate dimensions
    unsigned int block_size = props.maxThreadsPerBlock / 4,
    grid_size = ANT_COUNT / block_size;

    update_ant_positions<<<grid_size, block_size>>>(goal, d_paths, d_weights, d_graph, d_random_numbers, start);

    grid_size = (nc * nc) / block_size;
    pheromone_evaporation<<<grid_size, block_size>>>(d_weights);

    grid_size = ANT_COUNT / block_size;
    path_lengths<<<grid_size, block_size>>>(goal, d_paths, d_path_lengths);

    // copy results back from the device
    cutilSafeCall(hipMemcpy(&(*weights), d_weights, weights_length, hipMemcpyDeviceToHost));
    cutilSafeCall(hipMemcpy(&paths_gpu, d_paths, paths_length, hipMemcpyDeviceToHost));



    //cutilSafeCall(hipMemcpy(&graph, d_graph, graph_length, hipMemcpyDeviceToHost));

    size_t h_path_lengths[ANT_COUNT];
    cutilSafeCall(hipMemcpy(h_path_lengths, d_path_lengths, path_lengths_length, hipMemcpyDeviceToHost));

    std::cout << std::endl;

    int winning_ants = 0;
    for(int pathIndex = 0; pathIndex < ANT_COUNT; pathIndex++){
        //If reached the goal
        if(h_path_lengths[pathIndex] < MAX_STEPS){
            winning_ants++;

            for(int i = 0; i < MAX_STEPS - 1; i++){
                if(paths_gpu[pathIndex][i] != goal){
                    std::cout << paths_gpu[pathIndex][i + 1] << " ";

                    (*weights)[paths_gpu[pathIndex][i]][paths_gpu[pathIndex][i + 1]] += TOTAL_PHEROMONE_FOR_TRAIL / h_path_lengths[pathIndex];
                    (*weights)[paths_gpu[pathIndex][i + 1]][paths_gpu[pathIndex][i]] += TOTAL_PHEROMONE_FOR_TRAIL / h_path_lengths[pathIndex];
                }
            }

            std::cout << std::endl;

        }
    }

    std::cout << std::endl;

    std::cout << "Winning Ants" << winning_ants << std::endl;

// free device memory
cutilSafeCall(hipFree(d_paths));
cutilSafeCall(hipFree(d_weights));
cutilSafeCall(hipFree(d_graph));
cutilSafeCall(hipFree(d_path_lengths));
cutilSafeCall(hipFree(d_random_numbers));

// free host memory
//delete[] h_path_lengths;

// clean up all resources in this process associated with the device
hipDeviceReset();
}
